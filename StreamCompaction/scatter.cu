#include "hip/hip_runtime.h"

#include "scatter.h"


int * Scatter(int * input, int size){

	int * Md;
	int * Pd;
	int bsize = size * sizeof(int);
	int * output = new int[size];
	//int * temp;

	hipMalloc(&Md, bsize); 
	hipMalloc(&Pd, bsize); 
	//hipMalloc(&temp, bsize);
	hipMemcpy(Md, input, bsize, hipMemcpyHostToDevice);
	if(size>block_size){
		printf("WARNING: Array Size Over Block Size!\n");
		return NULL;
	}

	int numBlocks = (int)ceil(size/(float)block_size);

	//GPU version scatter
	GPUScatter<<<numBlocks, block_size>>> (Md, Pd, size);

	hipMemcpy(output, Pd, bsize, hipMemcpyDeviceToHost); 
	hipFree(Md);
	hipFree(Pd);

	return output;

}


__global__ void GPUScatter(int *in, int *out, int n){

	int thid = blockDim.x * blockIdx.x + threadIdx.x;

	if(thid<n){
		if(in[thid]!=0)
			out[thid] = 1;
		else
			out[thid] = 0;
	}
}
