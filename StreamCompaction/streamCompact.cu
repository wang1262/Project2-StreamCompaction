#include "hip/hip_runtime.h"
#include "streamCompact.h"
#include "scatter.h"

int * ThrustStreamCompact(int * input, int size, int value){

	int * output = new int[size];
	//thrust::copy_if(input, input[size], output, is_not_zero());
	return output;
}

int * StreamCompact(int * input, int * bool_input, int size, int cnt){

	int * Md;
	int * Md2;
	int * Pd;
	int bsize = size * sizeof(int);
	int * output = new int[cnt];
	//int * temp;

	hipMalloc(&Md, bsize); 
	hipMalloc(&Md2, bsize); 
	hipMalloc(&Pd, cnt*sizeof(int)); 
	//hipMalloc(&temp, bsize);
	hipMemcpy(Md, input, bsize, hipMemcpyHostToDevice);
	hipMemcpy(Md2, bool_input, bsize, hipMemcpyHostToDevice);

	int numBlocks = (int)ceil(size/(float)block_size);

	//GPU version scatter
	GPUStreamCompact<<<numBlocks, block_size>>> (Md, Md2, Pd, size);

	hipMemcpy(output, Pd, bsize, hipMemcpyDeviceToHost);
	hipFree(Md);
	hipFree(Md2);
	hipFree(Pd);

	return output;

}


__global__ void GPUStreamCompact(int *in, int * in2, int *out, int n){

	int thid = blockDim.x * blockIdx.x + threadIdx.x;
	if(thid<n){
		if(in2[thid]==1)
			out[thid] = in[thid];
	}
}