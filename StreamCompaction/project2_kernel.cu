#include "hip/hip_runtime.h"
#include "project2_kernel.h"
#include "prefixSum.h"
#include "scatter.h"
#include "streamCompact.h"

using namespace std;


int * randomIntArray(int size){
	int * M = new int[size];
	for(int i=0; i<size; i++){
		M[i] = rand()%10;
		//printf("orig_array[%d] = %d\n", i, M[i]);
	}
	return M;
}

void printOutput(int* arr, int size){

	for(int i=0; i<size; i++)
		printf("output[%d] = %d\n", i, arr[i]);
}


void project2_kernel()
{

	//initilize the input array
	int * input = new int[array_size];
	input = randomIntArray(array_size);
	printf("The input array generated below:\n");
	printOutput(input, array_size);


	//part1.....
	int * output = new int[array_size];
	printf("\nPart1 CPU prefix Sum.....\n");
	clock_t t1,t2;
	t1=clock();
	output = serialSum(input, array_size);
	t2=clock(); 
	double diff = ((double)t2-(double)t1);
	//double diffms=(diff)/(CLOCKS_PER_SEC/1000);
	printf("The CPU serial version of sum cost: %f ms.\n", diff);

	//printOutput(output, array_size);


	//part2.....
	printf("\n\nPart2 GPU prefix Sum.....\n");
	int * output2 = new int[array_size];
	t1=clock();
	output2 = NaivePrefixSum(input, array_size);
	t2=clock(); 
	diff = ((double)t2-(double)t1);
	//diffms=(diff)/(CLOCKS_PER_SEC/1000);
	printf("The GPU Naive prefix sum cost: %f ms.\n", diff);

	//printOutput(output2, array_size);


	//Part3a.....
	printf("\n\nPart3a GPU prefix Sum with Shared Memory on one block.....\n");
	int * output3 = new int[array_size];
	t1=clock();
	output3 = AdvancedPrefixSum(input, array_size);
	t2=clock(); 
	diff = ((double)t2-(double)t1);
	//diffms=(diff)/(CLOCKS_PER_SEC/1000);
	printf("The GPU Advanced prefix sum cost on single block: %f ms.\n", diff);

	//printOutput(output3, array_size);


	//Part3b.....
	printf("\n\nPart3b GPU prefix Sum with Shared Memory of Arbitrary Length.....\n");
	int * output4 = new int[array_size];
	t1=clock();
	output4 = AdvancedPrefixSumArbiLength(input, array_size);
	t2=clock(); 
	diff = ((double)t2-(double)t1);
	//diffms=(diff)/(CLOCKS_PER_SEC/1000);
	printf("The GPU Advanced prefix sum cost with arbitrary length: %f ms.\n", diff);

	//printOutput(output4, array_size);


	//Part4.....
	printf("\n\nPart4 GPU Scatter.....\n");
	int * output5_bool = new int[array_size];
	t1=clock();
	output5_bool = Scatter(input, array_size);
	t2=clock(); 
	diff = ((double)t2-(double)t1);
	//diffms=(diff)/(CLOCKS_PER_SEC/1000);
	printf("The GPU Scatter cost: %f ms.\n", diff);
	//printOutput(output5_bool, array_size);

	int cnt = 0;
	for(int i=0; i<array_size; i++)
		if(output5_bool[i]==1) 
			cnt++;

	printf("\n\n(Cont.)Stream Compact.....\n");
	int * output5 = new int[cnt];
	t1=clock();
	output5 = StreamCompact(input, output5_bool, array_size, cnt);
	t2=clock(); 
	diff = ((double)t2-(double)t1);
	//diffms=(diff)/(CLOCKS_PER_SEC/1000);
	printf("The GPU Scatter cost: %f ms.\n", diff);
	//printOutput(output5, cnt);
	

	printf("\n\n(Cont.)Thrust Version Stream Compact.....\n");
	int * output6 = new int[cnt];
	t1=clock();
	output6 = ThrustStreamCompact(input, cnt, 1);
	t2=clock(); 
	diff = ((double)t2-(double)t1);
	//diffms=(diff)/(CLOCKS_PER_SEC/1000);
	printf("The GPU Scatter cost: %f ms.\n", diff);
	//printOutput(output6, cnt);


	printf("\n\n<-----Work Done!----->\n\n");

}