#include "hip/hip_runtime.h"
#include "prefixSum.h"


__global__ void GPUAdvancedArbitraryLength(int * in, int * blocksum, int * out, int n){

	//allocate shared memory of this single block
	__shared__ int Md[block_size];
	__shared__ int Pd[block_size];

	int thid = blockDim.x * blockIdx.x + threadIdx.x;
	int cur_thidx = threadIdx.x;

	if(thid<n){

		Md[cur_thidx] = in[thid];
		__syncthreads();

		//after the input array been loaded into shared memory then do the Sum
		int d_max = (int)ceil(log2((float)n));
		for(int d=1; d<=d_max; d++){
			if(cur_thidx>= (int)pow(2.0, d-1))
				Pd[cur_thidx] = Md[cur_thidx - (int)pow(2.0, d-1)] + Md[cur_thidx];
			else
				Pd[cur_thidx] = Md[cur_thidx];
			Md[cur_thidx] = Pd[cur_thidx];
			__syncthreads();
		}

		if(cur_thidx == block_size-1)
			blocksum[blockIdx.x] = Pd[cur_thidx];

		out[thid] = Pd[cur_thidx];
	}
}

__global__ void AddSum(int * input, int * output){

	int thid = blockDim.x * blockIdx.x + threadIdx.x;
	//add the sum from all blocks to final output
	int blockSumIndex = blockIdx.x - 1;

	if(blockSumIndex!=0)
	    output[thid] += input[blockSumIndex];

}

int * AdvancedPrefixSumArbiLength(int * input, int size){

	int * Md;
	int * Pd;
	int * blockSum;
	int * finalValue;
	int bsize = size * sizeof(int);
	int * output = new int[size];
	//int * temp;
	int numBlocks = (int)ceil(size/(float)block_size);
	//calculate how many blocks needed for the sum

	hipMalloc(&Md, bsize); 
	hipMalloc(&Pd, bsize); 
	hipMalloc(&blockSum, sizeof(int)*numBlocks);
	//hipMalloc(&temp, bsize);
	hipMemcpy(Md, input, bsize, hipMemcpyHostToDevice);

	GPUAdvancedArbitraryLength<<<numBlocks, block_size>>> (Md, blockSum, Pd, size);

	//Perform scan independantly on each chunch while storing the total sum in a new array sums
	if(numBlocks>1)
	{
		int numBlocks2 = (int)ceil(numBlocks/(float)block_size);
		for(int d=1; (int)pow(2.0,d-1)<=numBlocks; d++)
		{
			GPUAdvancedSingleBlock<<<numBlocks2, block_size>>> (blockSum, finalValue, size);
			
			std::swap(finalValue, blockSum);
		}

		//Add up SUM value from each block
		AddSum<<<numBlocks, block_size>>> (finalValue, Pd);
	}


	output[0] = 0;
	hipMemcpy(&output[1], Pd, bsize, hipMemcpyDeviceToHost); 
	hipFree(Md);
	hipFree(Pd);

	return output;

}


__global__ void GPUAdvancedSingleBlock(int *in, int *out, int n){

	if(n>1000){
		//printf("WARNING: BUFFER SIZE OVER SHARED MEMORY LIMIT!\n");
		return;
	}

	//allocate shared memory of this single block
	__shared__ int Md[block_size];
	__shared__ int Pd[block_size];

	int thid = threadIdx.x;
	if(thid<n){

		Md[thid] = in[thid];

		__syncthreads();
		//after the input array been loaded into shared memory then do the Sum
		int d_max = (int)ceil(log2((float)n));
		for(int d=1; d<=d_max; d++){
			if(thid>= (int)pow(2.0, d-1))
				Pd[thid] = Md[thid - (int)pow(2.0, d-1)] + Md[thid];
			else
				Pd[thid] = Md[thid];
			Md[thid] = Pd[thid];
			__syncthreads();
		}	

		out[thid] = Pd[thid];
	}
}

int * AdvancedPrefixSum(int * input, int size){

	int * Md;
	int * Pd;
	int bsize = size * sizeof(int);
	int * output = new int[size];
	//int * temp;

	hipMalloc(&Md, bsize); 
	hipMalloc(&Pd, bsize); 
	//hipMalloc(&temp, bsize);
	hipMemcpy(Md, input, bsize, hipMemcpyHostToDevice);
	if(size>block_size){
		printf("WARNING: Array Size Over Block Size!\n");
		return NULL;
	}

	//single block
	int numBlocks = 1;
	//Double buffer version of sum scan
	GPUAdvancedSingleBlock<<<numBlocks, block_size>>> (Md, Pd, size);

	output[0] = 0;
	hipMemcpy(&output[1], Pd, bsize, hipMemcpyDeviceToHost); 
	hipFree(Md);
	hipFree(Pd);

	return output;

}


__global__ void GPUNaiveDoubleBuffer(int *in, int *out, int n, int d)
{
	int thid = blockDim.x * blockIdx.x + threadIdx.x;

	if(thid < n)
	{
		if(thid>= (int)pow(2.0, d-1))
			out[thid] = in[thid - (int)pow(2.0, d-1)] + in[thid];
		else
			out[thid] = in[thid];
	}	
}

int * NaivePrefixSum(int * input, int size){

	int * Md;
	int * Pd;
	int bsize = size * sizeof(int);
	int * output = new int[size];
	//int * temp;

	hipMalloc(&Md, bsize); 
	hipMalloc(&Pd, bsize); 
	//hipMalloc(&temp, bsize);
	hipMemcpy(Md, input, bsize, hipMemcpyHostToDevice);
	int numBlocks = (int)ceil(size/(float)block_size);

	int d_max = (int)ceil(log2((float)size));

	for(int d=1; d<=d_max; d++){
		//Double buffer version of sum scan
		GPUNaiveDoubleBuffer<<<numBlocks, block_size>>> (Md, Pd, size, d);
		std::swap(Md, Pd);
	}

	output[0] = 0;
	hipMemcpy(&output[1], Pd, bsize, hipMemcpyDeviceToHost); 
	hipFree(Md);
	hipFree(Pd);

	return output;
}

